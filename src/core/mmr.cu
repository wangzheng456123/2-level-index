#include <core/mmr.cuh>

void parafilter_mmr::init_mmr() {
    int device_count;
    hipGetDeviceCount(&device_count);

    total.resize(device_count);
    available.resize(device_count);
    cur_mems.resize(device_count);
    cur_offset.resize(device_count);
}

void* parafilter_mmr::mem_allocator(uint64_t size) 
{
    int id;
    hipGetDevice(&id);
    LOG(INFO) << "allocate memory with " << size << " byte on device :" << id; 
    if (id < cur_mems.size() && cur_mems[id].count(size) && cur_offset[id][size] < cur_mems[id][size].size()) {
      int offset = cur_offset[id][size];
      cur_offset[id][size]++;
      LOG(INFO) << "parafilter mmr allocate block from pool";
      return cur_mems[id][size][offset];
    }
    else {
      void* mem;
      hipMalloc((void**)&mem, size);
      LOG(INFO) << "parafilter mmr allocate block runtime";
      checkCUDAErrorWithLine("hipMalloc failed");
      workspace_add_mem(mem, size);
      cur_offset[id][size]++;
      LOG(INFO) << cur_mems[id][size].size() << " blocks, " << cur_offset[id][size]
        << " block offset";
      return mem;
    }
}

void parafilter_mmr::free_cur_workspace_device_mems(bool free_mems) 
{
    int id;
    hipGetDevice(&id);
    for (auto iter = cur_offset[id].begin(); iter != cur_offset[id].end(); ++iter) {
      iter->second = 0;
    }
    if (free_mems) {
      for (auto iter = cur_mems[id].begin(); iter != cur_mems[id].end(); ++iter) {
        auto key = iter->first;
        for (auto _iter = cur_mems[id][key].begin(); _iter != cur_mems[id][key].end(); ++_iter) { 
            LOG(INFO) << "free device mem with" << *_iter;
            hipFree(*_iter);
            checkCUDAErrorWithLine("free work space memory failed");
        }
        cur_mems[id][key].clear();
      }
      cur_mems[id].clear();
    }
}




